#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include <sys/time.h>
#include <sys/types.h>
#include <math.h>

//#define DIM 32
//#define DATA_TYPE float
//#define NTHREADS 16 //max 1024
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_NUM 10

void Malloc2d(DATA_TYPE ***a, int dimx, int dimy) {
        *a = (DATA_TYPE**)malloc(dimx * sizeof(DATA_TYPE*));
        if (*a == NULL){
                printf("ERROR: out of memory\n");
        };
        for (int i=0; i<dimx; i++) {
                (*a)[i] = (DATA_TYPE*)malloc(dimy * sizeof(DATA_TYPE));
                if ((*a)[i] == NULL){
                        printf("ERROR: out of memory\n");
                };
        };
};

void Init1d(DATA_TYPE *a, DATA_TYPE *a_ref) {
        int i,j;
        for(i=0; i<DIM; ++i)
         	 for(j=0; j<DIM; ++j) {
                a[i*DIM+j] = (DATA_TYPE)rand()/(DATA_TYPE)(RAND_MAX/MAX_NUM);
                a_ref[i*DIM+j] = a[i*DIM+j];
            };
};

void PrintMatrix(DATA_TYPE *a, int xdim) {
	int i,j;
	for (i=0; i<xdim; i++) {
		for (j=0; j<DIM; j++) {
			printf("%f ", a[i*DIM+j]);
		};
		printf("\n");
	};
};

__device__  int ModifyRows(DATA_TYPE *a, int start_row, int end_row, int col, DATA_TYPE *b) {
#ifdef HIGH_VERB	
	printf("ModifyRows %d-%d with row %d\n", start_row, end_row, col);
#endif	
	int i,k;
	for (i=start_row; i<=end_row; i++) {
		if (a[i*DIM+col]==0) {
			continue;
		};
		if (b[col]==0) {
			printf("a[%d][%d]=0!!!", i, col);
			return 1;
		};
		__shared__ DATA_TYPE coeff;
		coeff = a[i*DIM+col]/b[col];
		for (k=col+1; k<DIM; k++) {
			a[i*DIM+k] -= b[k] * coeff;
		};
		a[i*DIM+col]=0;
#ifdef HIGH_VERB
		printf("row%d = row%d - row%d * %f\n",i, i, col, coeff);
		for (k=0; k<DIM; k++) {
			printf("%f ", a[i*DIM+k]);
		};
		printf("\n");
#endif	
	};
	
	return 0;
};

__global__ void ModifyMatrix(DATA_TYPE *a, int step) {
    int offset = MAX_THREADS_PER_BLOCK*blockIdx.x + step*threadIdx.x;
    int j,i;
    //__shared__ DATA_TYPE pivot_row[DIM];
	//printf("ModifyMatrix called offset=%d bx=%d tx=%d step=%d\n", offset, blockIdx.x, threadIdx.x, step);
	for (j=0; j<DIM; j++) {
		//for (i=offset; i<offset+step; i++) {
		//	pivot_row[i]=a[j*DIM+i];
		//}
		__syncthreads();
		if (j<(offset+step)) {
#ifdef HIGH_VERB		
			printf("task=%d Modifying all rows using pivot from row %d\n", offset, j); 
#endif	
			int start_row = j>=offset? j+1 : offset;
			if (ModifyRows(a, start_row, offset+step-1, j, &a[j*DIM])) {
				break;
			}	
		};	
		__syncthreads();
	};
	for (j=offset+1; j<offset+step; j++) {
#ifdef HIGH_VERB		
		printf("task=%d Modifying all rows using pivot from row %d\n", thread_id, j-1); 
#endif		
		if (ModifyRows(a, j, offset+step-1, j-1, &a[(j-1)*DIM+0])) {
			break;
		};
	__syncthreads();
	};
}

void sequential_gaussian(DATA_TYPE *a) {
	int i,j,k;
	for (j=1; j<DIM; j++) {

		int col = j-1;
		for (i=j; i<DIM; i++) {

			if (a[i*DIM+col]==0) {
				continue;
			};
			DATA_TYPE coeff = a[i*DIM+col]/a[col*DIM+col];
						
			for (k=col+1; k<DIM; k++) {
				a[i*DIM+k] -= a[col*DIM+k] * coeff;
			};
			a[i*DIM+col] = 0;
		};
	};
}

int main() {
	//int i;
	srand(time(NULL));
	DATA_TYPE *a = (DATA_TYPE*)malloc(DIM*DIM*sizeof(DATA_TYPE));
	DATA_TYPE *a_ref = (DATA_TYPE*)malloc(DIM*DIM*sizeof(DATA_TYPE));
	Init1d(a, a_ref);
#ifdef PRINT	
	printf("Matrix before: \n");
	PrintMatrix(a, DIM);
#endif	
	DATA_TYPE *a_gpu;
	hipMalloc((void **)&a_gpu, DIM*DIM*sizeof(DATA_TYPE));
	hipMemcpy(a_gpu, a, DIM*DIM*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	struct timeval start, stop;
	dim3 dimGrid (1, 1, 1);
  	dim3 dimBlock(NTHREADS, 1, 1);
	printf("starting to measure runtime\n");
	gettimeofday(&start, 0);
	ModifyMatrix<<<dimGrid,dimBlock>>> (a_gpu, DIM/NTHREADS);
	hipDeviceSynchronize();
	gettimeofday(&stop, 0);
	printf("Time = %.6f\n", (stop.tv_sec+stop.tv_usec*1e-6)-(start.tv_sec+start.tv_usec*1e-6));
	printf("copying from gpu to cpu\n");
	hipMemcpy(a, a_gpu, DIM*DIM*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("freeing the gpu mem\n");
	hipFree(a_gpu);
#ifdef PRINT
	printf("Matrix after: \n");
	PrintMatrix(a, DIM);
#endif
#ifdef VERIFY
	sequential_gaussian(a_ref);
#ifdef PRINT
	printf("Matrix verify: \n");
	PrintMatrix(a_ref, DIM);
#endif
	int rand_x = (int)rand()%DIM;
	int rand_y = (int)rand()%DIM;
	if (a[rand_x*DIM+rand_y] != a_ref[rand_x*DIM+rand_y]) {
		printf("a[%d][%d] (%f) != a_ref[%d][%d] (%f)", rand_x, rand_y, a[rand_x*DIM+rand_y], rand_x, rand_y, a_ref[rand_x*DIM+rand_y]);
		return 0;
	};
#endif
	return 0;
}


