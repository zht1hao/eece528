#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}
int main(){
	int a,b,c;
	int *d_a,*d_b,*d_c;	
	int size = sizeof(int);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	a = 2;
	b = 7;
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);
	
	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
	
	printf("c = %d\n",c);	
	printf("a+b = %d\n",a+b);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}